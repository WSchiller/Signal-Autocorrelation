#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE		64     // number of threads per block
#endif

#define	SIZE		32768      // data set size of signals.txt file

#define NUMBLOCKS		( SIZE / BLOCKSIZE )  // Number of thread blocks 

// Global host arrays
float hA[SIZE * 2];
float hSums[SIZE];


// CUDA kernel function
__global__  void shiftMultiply(float* dA, float* dSums)
{
	unsigned int wgNumber = blockIdx.x;
	unsigned int wgDimension = blockDim.x;
	unsigned int threadNum = threadIdx.x;
	unsigned int gid = wgNumber * wgDimension + threadNum;  // global id of thread

	// calculate the shift sums for each signal value
	int shift = gid;
	float sum = 0.;
	for (int i = 0; i < SIZE; i++)
	{
		sum += dA[i] * dA[i + shift];
	}
	dSums[shift] = sum;
	
}


int main() {

	// File pointer to read text file
	FILE* fp = fopen("signal.txt", "r");

	if (fp == NULL)
	{
		fprintf(stderr, "Cannot open file 'signal.txt'\n");
		exit(1);
	}

	int Size;
	fscanf(fp, "%d", &Size);
	for (int i = 0; i < Size; i++)
	{
		fscanf(fp, "%f", &hA[i]);
		hA[i + Size] = hA[i];		// fill 2nd half of array
	}
	fclose(fp);

	// allocate device memory:
	float* dA, * dSums;
	
	dim3 dimsA(SIZE * 2, 1, 1);
	dim3 dimsSums(SIZE, 1, 1);

	hipError_t status;
	status = hipMalloc((void**)(&dA), sizeof(hA));
	checkCudaErrors(status);

	status = hipMalloc((void**)(&dSums), sizeof(hSums));
	checkCudaErrors(status);
	
	// copy host memory to the device:
	status = hipMemcpy(dA, hA, SIZE*2*sizeof(float), hipMemcpyHostToDevice);
	checkCudaErrors(status);

	status = hipMemcpy(dSums, hSums, SIZE*sizeof(float), hipMemcpyHostToDevice);
	checkCudaErrors(status);

	// setup the execution parameters:
	dim3 grid(NUMBLOCKS, 1, 1);
	dim3 threads(BLOCKSIZE, 1, 1);
	
	// create and start timer
	hipDeviceSynchronize();

	// allocate CUDA events that we'll use for timing:
	hipEvent_t start, stop;
	status = hipEventCreate(&start);
	checkCudaErrors(status);
	status = hipEventCreate(&stop);
	checkCudaErrors(status);

	// record the start event:
	status = hipEventRecord(start, NULL);
	checkCudaErrors(status);

	// execute the kernel:
	shiftMultiply << < grid, threads >> > (dA, dSums);
	
	// record the stop event:
	status = hipEventRecord(stop, NULL);
	checkCudaErrors(status);

	// wait for the stop event to complete:
	status = hipEventSynchronize(stop);
	checkCudaErrors(status);

	float msecTotal = 0.0f;
	status = hipEventElapsedTime(&msecTotal, start, stop);
	checkCudaErrors(status);

	// compute and print the performance
	double secondsTotal = 0.001 * (double)msecTotal;
	double MultsPerSecond = (float)SIZE * (float)SIZE / secondsTotal;
	double MegaMultsPerSecond = MultsPerSecond / 1000000.;
	fprintf(stderr, "Number of Threads Per Block: %d\n", BLOCKSIZE);
	fprintf(stderr, "Array Size = %10d\tMegaMults/Second = %10.2lf\n", SIZE, MegaMultsPerSecond);

	// copy result from the device to the host:
	status = hipMemcpy(hSums, dSums, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	checkCudaErrors(status);
	hipDeviceSynchronize();

	// Print the sums from host device, Sums[1]...Sums[512]
	for(int i = 1; i <= 512 ; i++) {
		printf("Shift Number: %d\tSum: %5.2lf\n", i, hSums[i]);
	}

	// clean up memory:
	status = hipFree(dA);
	status = hipFree(dSums);
	checkCudaErrors(status);
	
	return 0;
}